#include "hip/hip_runtime.h"
#include "lib/Common.h"
#include "lib/Model.h"
#include "KFusionTracker.h"

const float3& reinterpretVec3AsFloat3(const glm::vec3& v)
{
    return *reinterpret_cast<const float3*>(&v);
}

__global__ void getCostForEachVertex(float* costs, float3* vertexData, Volume volume, Matrix4 transform)
{
    int index = threadIdx.x;

    // Transform vertex
    float3 vertex = transform * vertexData[index];

    // Calculate cell position in the volume and check bounds
    float trunc = 0.5f;
    int3 scaledPos = make_int3(
        vertex.x * volume.size.x / volume.dim.x,
        vertex.y * volume.size.y / volume.dim.y,
        vertex.z * volume.size.z / volume.dim.z);
    if (scaledPos.x >= 0 && scaledPos.y >= 0 && scaledPos.z >= 0 &&
        scaledPos.x < volume.size.x && scaledPos.y < volume.size.y && scaledPos.z < volume.size.z)
    {
        costs[index] = fmin(volume.interp(vertex), trunc);
    }
    else
    {
        costs[index] = trunc;
    }
}

#define COUNT 16

float getCost(Model* model, Volume volume, const glm::mat4& transform)
{
    // Select some vertices
    // TODO: Make this work for vertex count < 512
    assert(model->getVertices().size() >= COUNT);

    // Allocate space
    float3* vertices = new float3[COUNT];
    float* costs = new float[COUNT];
    float3* deviceVertices;
    float* deviceCosts;
    hipMalloc(&deviceVertices, sizeof(float3) * COUNT);
    hipMalloc(&deviceCosts, sizeof(float) * COUNT);

    // TODO: Randomly select these vertices
    for (int i = 0; i < COUNT; i++)
    {
        const glm::vec3& srcVertex = model->getVertices()[i];
        vertices[i].x = srcVertex.x;
        vertices[i].y = srcVertex.y;
        vertices[i].z = srcVertex.z;
    }

    // Copy to GPU memory
    CUDA_CHECK(hipMemcpy(deviceVertices, vertices, sizeof(float3) * COUNT, hipMemcpyHostToDevice));

    // Call kernel
    getCostForEachVertex<<<1, COUNT>>>(deviceCosts, deviceVertices, volume, glmToKFusion(transform));
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy and sum results
    CUDA_CHECK(hipMemcpy(costs, deviceCosts, sizeof(float) * COUNT, hipMemcpyDeviceToHost));
    float sum = 0.0f;
    for (int i = 0; i < COUNT; i++)
        sum += costs[i];

    // Free memory and return
    delete[] vertices;
    delete[] costs;
    hipFree(deviceVertices);
    hipFree(deviceCosts);
    return sum;
}
